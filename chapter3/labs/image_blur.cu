
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#define cudaCheckError()                                                                     \
    {                                                                                        \
        hipError_t e = hipGetLastError();                                                  \
        if (e != hipSuccess)                                                                \
        {                                                                                    \
            printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
            exit(EXIT_FAILURE);                                                              \
        }                                                                                    \
    }

#define BLUR_SIZE 1

__global__
void blur_kernel(unsigned char* in, unsigned char* out, int width, int height){

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    //if(col >= width || row >= height)
    //    printf("%d %d\n %d %d\n %d %d\n %d %d\n\n", col, row, blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, threadIdx.x, threadIdx.y);

    if(col >= width || row >= height) return;

    int pixVal = 0;
    int pixels = 0;

    // Computer the average of the neighboring pixels
    for(int blurrow = -BLUR_SIZE; blurrow < BLUR_SIZE + 1; ++blurrow){
        for(int blurcol = -BLUR_SIZE; blurcol < BLUR_SIZE + 1; ++blurcol){
            int currow = row + blurrow;
            int curcol = col + blurcol;

            // Check if the current pixel is in the image
            if(curcol < 0 || curcol >= width || currow < 0 || currow >= height) continue;

            pixVal += in[currow * width + curcol];
            ++pixels;// Count the number of pixel values that have been added
        }
    }
    // printf("%d %d\n", pixVal, pixels);
    // Write out the result for this pixel
    out[row * width + col] = (unsigned char) ((float)pixVal / pixels);
}


int main(int argc, char* argv[]){

    int block_size;
    int image_width;
    int image_height;
    int size;
    unsigned char* h_input_image, *h_output_image;
    unsigned char* d_input_image, *d_output_image;

    if(argc != 2){
        printf("Usage: %s <block_size>\n", argv[0]);
        return 1;
    }
    block_size = strtol(argv[1], NULL, 10);

    // Set the width and height of the image
    image_width = 50;
    image_height = 20;
    size = image_width * image_height;

    // Allocate memory for the input and output images and initialize the input image
    h_input_image = (unsigned char*) malloc(size * sizeof(unsigned char));
    h_output_image = (unsigned char*) malloc(size * sizeof(unsigned char));
    for(int i = 0; i < size; ++i)
        h_input_image[i] = rand() % 256;

    // Show the input image
    for(int i = 0; i < image_height; ++i){
        for(int j = 0; j < image_width; ++j)
            printf("%d ", h_input_image[i * image_width + j]);
        printf("\n");
    }
    printf("\n");

    // Copy the input image to the device
    hipMalloc((void**) &d_input_image, size * sizeof(unsigned char));
    hipMalloc((void**) &d_output_image, size * sizeof(unsigned char));
    hipMemcpy(d_input_image, h_input_image, size * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Blur the image
    dim3 dimGrid(ceil((float)image_width/block_size), ceil((float)image_height/block_size), 1);
    dim3 dimBlock(block_size, block_size, 1);
    //printf("%d %d\n", dimGrid.x, dimGrid.y);
    //printf("%d %d\n", dimBlock.x, dimBlock.y);
    //printf("\n");
    blur_kernel<<<dimGrid, dimBlock>>>(d_input_image, d_output_image, image_width, image_height);

    // Copy the output back to the host
    hipMemcpy(h_output_image, d_output_image, size * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Show the output image
    for(int i = 0; i < image_height; ++i){
        for(int j = 0; j < image_width; ++j)
            printf("%d ", h_output_image[i * image_width + j]);
        printf("\n");
    }

    // Free the device memory
    hipFree(d_input_image);
    hipFree(d_output_image);

    // Free the host memory
    free(h_input_image);
    free(h_output_image);
}