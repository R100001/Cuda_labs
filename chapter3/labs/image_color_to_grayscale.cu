
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#define cudaCheckError()                                                                     \
    {                                                                                        \
        hipError_t e = hipGetLastError();                                                  \
        if (e != hipSuccess)                                                                \
        {                                                                                    \
            printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
            exit(EXIT_FAILURE);                                                              \
        }                                                                                    \
    }

#define NUM_CHANNELS 3

__global__
void color_to_grayscale_conversion(unsigned char* in, unsigned char* out, int width, int height){

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < 0 || row >= height || col < 0 || col >= width) return;

    int grey_offset = row * width + col;

    int rgb_offset = grey_offset * NUM_CHANNELS;

    unsigned char r = in[rgb_offset + 0];
    unsigned char g = in[rgb_offset + 1];
    unsigned char b = in[rgb_offset + 2];

    out[grey_offset] = (unsigned char)(0.21f * r + 0.71f * g + 0.07f * b);
}


int main(int argc, char* argv[]){

    int block_dim_x, block_dim_y;
    int image_width;
    int image_height;
    int size;
    unsigned char* h_input_image, *h_output_image;
    unsigned char* d_input_image, *d_output_image;

    if(argc != 3){
        printf("Usage: %s <block_dim_x> <block_dim_y>\n", argv[0]);
        return 1;
    }
    block_dim_x = strtol(argv[1], NULL, 10);
    block_dim_y = strtol(argv[2], NULL, 10);
    if(block_dim_x <= 0 || block_dim_y <= 0){
        printf("Block dimensions must be positive\n");
        return 1;
    }
    if(block_dim_x * block_dim_y > 1024){
        printf("block_dim_x * block_dim_y must be less than 1024\n");
        return 1;
    }

    // Set the width and height of the image
    image_width = 50;
    image_height = 20;
    size = image_width * image_height;

    // Allocate memory for the input and output images and initialize the input image
    h_input_image = (unsigned char*) malloc(NUM_CHANNELS * size * sizeof(unsigned char));
    h_output_image = (unsigned char*) malloc(size * sizeof(unsigned char));
    for(int i = 0; i < NUM_CHANNELS * size; ++i)
        h_input_image[i] = rand() % 256;

    // Show the input image
    for(int c = 0; c < NUM_CHANNELS; ++c){
        for(int i = 0; i < image_height; ++i){
            for(int j = 0; j < image_width; ++j){
                printf("%d ", h_input_image[NUM_CHANNELS * (i * image_width + j) + c]);
            }
            printf("\n");
        }
        printf("\n");
    }
    printf("\n");

    // Copy the input image to the device
    hipMalloc((void**) &d_input_image, NUM_CHANNELS * size * sizeof(unsigned char));
    hipMalloc((void**) &d_output_image, size * sizeof(unsigned char));
    hipMemcpy(d_input_image, h_input_image, NUM_CHANNELS * size * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Blur the image
    dim3 dimGrid(ceil((float)image_width/block_dim_x), ceil((float)image_height/block_dim_y), 1);
    dim3 dimBlock(block_dim_x, block_dim_y, 1);
    //printf("%d %d\n", dimGrid.x, dimGrid.y);
    //printf("%d %d\n", dimBlock.x, dimBlock.y);
    //printf("\n");
    color_to_grayscale_conversion<<<dimGrid, dimBlock>>>(d_input_image, d_output_image, image_width, image_height);

    // Copy the output back to the host
    hipMemcpy(h_output_image, d_output_image, size * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Show the output image
    for(int i = 0; i < image_height; ++i){
        for(int j = 0; j < image_width; ++j)
            printf("%d ", h_output_image[i * image_width + j]);
        printf("\n");
    }

    // Free the device memory
    hipFree(d_input_image);
    hipFree(d_output_image);

    // Free the host memory
    free(h_input_image);
    free(h_output_image);

    cudaCheckError();
}