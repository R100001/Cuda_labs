
#include <hip/hip_runtime.h>
#include <iostream>

int main()
{

    int dev_count;
    hipGetDeviceCount(&dev_count);

    for (int i = 0; i < dev_count; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << "Device " << i << ": " << prop.name << "\n";
        std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << "\n";
        std::cout << "Max threads per SM: " << prop.maxThreadsPerMultiProcessor << "\n";
        std::cout << "Number of SMs: " << prop.multiProcessorCount << "\n";
        std::cout << "Warp size: " << prop.warpSize << "\n";
        std::cout << "Max threads on block's x dim: " << prop.maxThreadsDim[0] << "\n";
        std::cout << "Max threads on block's y dim: " << prop.maxThreadsDim[1] << "\n";
        std::cout << "Max threads on block's z dim: " << prop.maxThreadsDim[2] << "\n";
        std::cout << "Max blocks on grid's x dim: " << prop.maxGridSize[0] << "\n";
        std::cout << "Max blocks on grid's y dim: " << prop.maxGridSize[1] << "\n";
        std::cout << "Max blocks on grid's z dim: " << prop.maxGridSize[2] << "\n";
    }



}